#include "hip/hip_runtime.h"
#pragma once

#include <thrust/device_ptr.h>
#include <thrust/device_malloc.h>
#include <thrust/device_vector.h>
#include <thrust/copy.h>

#include "fmmtl/config.hpp"

#include "fmmtl/dispatch/S2T/S2T_Compressed.hpp"
#include "fmmtl/dispatch/S2T/S2T_Blocked_CSR.cu"

struct Data {
  unsigned num_sources;
  unsigned num_targets;
  unsigned num_blocks;
  Data(unsigned s, unsigned t, unsigned b)
      : num_sources(s),
        num_targets(t),
        num_blocks(b) {
  }
};


template <typename T>
inline T* gpu_new(unsigned n) {
  return thrust::raw_pointer_cast(thrust::device_malloc<T>(n));
}

template <typename Container>
inline typename Container::value_type* gpu_copy(const Container& c) {
  typedef typename Container::value_type c_value;
  // Allocate
  thrust::device_ptr<c_value> dptr = thrust::device_malloc<c_value>(c.size());
  // Copy
  //thrust::uninitialized_copy(c.begin(), c.end(), dptr);
  thrust::copy(c.begin(), c.end(), dptr);
  // Return
  return thrust::raw_pointer_cast(dptr);
}

template <typename T>
inline void gpu_free(T* p) {
  thrust::device_free(thrust::device_pointer_cast<void>(p));
}

template <typename Kernel>
S2T_Compressed<Kernel>::S2T_Compressed()
    : data_(0) {
}

template <typename Kernel>
S2T_Compressed<Kernel>::S2T_Compressed(
    std::vector<std::pair<unsigned,unsigned> >& target_ranges,
    std::vector<unsigned>& source_range_ptrs,
    std::vector<std::pair<unsigned,unsigned> >& source_ranges,
    const std::vector<source_type>& sources,
    const std::vector<target_type>& targets)
    : data_(new Data(sources.size(), targets.size(), target_ranges.size())),
      target_ranges_(gpu_copy(target_ranges)),
      source_range_ptrs_(gpu_copy(source_range_ptrs)),
      source_ranges_(gpu_copy(source_ranges)),
      sources_(gpu_copy(sources)),
      targets_(gpu_copy(targets)) {
}

template <typename Kernel>
S2T_Compressed<Kernel>::~S2T_Compressed() {
  delete reinterpret_cast<Data*>(data_);
  gpu_free(target_ranges_);
  gpu_free(source_range_ptrs_);
  gpu_free(source_ranges_);
  gpu_free(sources_);
  gpu_free(targets_);
}

/** A functor that indexes an array as one type but returns another type */
template <typename T1, typename T2>
class tricky_cast {
  T1* a_;
 public:
  __host__ __device__
  tricky_cast(T1* a) : a_(a) {}
  __host__ __device__
  T2 operator[](unsigned blockidx) const {
    return *((T2*)(a_ + blockidx));
  }
};

template <typename Kernel>
void S2T_Compressed<Kernel>::execute(
    const Kernel& K,
    const std::vector<charge_type>& charges,
    std::vector<result_type>& results) {
  typedef Kernel kernel_type;
  typedef typename kernel_type::source_type source_type;
  typedef typename kernel_type::target_type target_type;
  typedef typename kernel_type::charge_type charge_type;
  typedef typename kernel_type::result_type result_type;

  // XXX: Using a device_vector here was giving "floating point exceptions"...
  // XXX: device_vector doesn't like the Vec?
  charge_type* d_charges = gpu_copy(charges);
  result_type* d_results = gpu_copy(results);

  Data* data = reinterpret_cast<Data*>(data_);

  // TODO: set tpb to ncrit
  const unsigned num_tpb    = 256;
  const unsigned num_blocks = data->num_blocks;

#if defined(FMMTL_DEBUG)
  std::cout << "Launching GPU Kernel: (blocks, threads/block) = ("
            << num_blocks << ", " << num_tpb << ")" << std::endl;
#endif

  typedef thrust::pair<unsigned,unsigned> upair;

  // Launch kernel <<<grid_size, block_size>>>
  blocked_p2p<num_tpb><<<num_blocks,num_tpb>>>(
      K,
      target_ranges_,
      tricky_cast<unsigned, upair>(source_range_ptrs_),
      source_ranges_,
      sources_,
      //thrust::raw_pointer_cast(d_charges.data()),
      d_charges,
      targets_,
      d_results);
      //thrust::raw_pointer_cast(d_results.data()));
  FMMTL_CUDA_CHECK;

  // Copy results back
  thrust::device_ptr<result_type> d_results_ptr = thrust::device_pointer_cast(d_results);
  thrust::copy(d_results_ptr, d_results_ptr + results.size(), results.begin());

  gpu_free(d_results);
  gpu_free(d_charges);
}


/** A functor that maps blockidx -> (target_begin,target_end) */
template <unsigned BLOCKDIM>
class block_range {
  unsigned N_;
 public:
  __host__ __device__
  block_range(unsigned N) : N_(N) {}
  __host__ __device__
  thrust::pair<unsigned,unsigned> operator[](unsigned blockidx) const {
    return thrust::make_pair(blockidx * BLOCKDIM,
                             min(blockidx * BLOCKDIM + BLOCKDIM, N_));
  }
};

/** A functor that returns a constant */
template <typename T>
class constant {
  T value_;
 public:
  __host__ __device__
  constant(T value) : value_(value) {}
  __host__ __device__
  T operator[](unsigned) const {
    return value_;
  }
};

template <typename Kernel>
void
S2T_Compressed<Kernel>::execute(const Kernel& K,
                                const std::vector<source_type>& s,
                                const std::vector<charge_type>& c,
                                const std::vector<target_type>& t,
                                std::vector<result_type>& r) {
  typedef Kernel kernel_type;
  typedef typename kernel_type::source_type source_type;
  typedef typename kernel_type::target_type target_type;
  typedef typename kernel_type::charge_type charge_type;
  typedef typename kernel_type::result_type result_type;

  source_type* d_sources = gpu_copy(s);
  charge_type* d_charges = gpu_copy(c);
  target_type* d_targets = gpu_copy(t);
  result_type* d_results = gpu_copy(r);

  // XXX: device_vector doesn't like our vector?
  //thrust::device_vector<source_type> d_sources(s);
  //thrust::device_vector<charge_type> d_charges(c);
  //thrust::device_vector<target_type> d_targets(t);
  //thrust::device_vector<result_type> d_results(r);

  const unsigned num_tpb    = 256;
  const unsigned num_blocks = (t.size() + num_tpb - 1) / num_tpb;

#if defined(FMMTL_DEBUG)
  std::cout << "Launching GPU Kernel: (blocks, threads/block) = ("
            << num_blocks << ", " << num_tpb << ")" << std::endl;
#endif

  typedef thrust::pair<unsigned,unsigned> upair;

  // Launch kernel <<<grid_size, block_size>>>
  blocked_p2p<num_tpb><<<num_blocks, num_tpb>>>(
      K,
      block_range<num_tpb>(t.size()),
      constant<upair>(upair(0,1)),
      constant<upair>(upair(0,s.size())),
      d_sources,
      d_charges,
      d_targets,
      d_results);
      //thrust::raw_pointer_cast(d_sources.data()),
      //thrust::raw_pointer_cast(d_charges.data()),
      //thrust::raw_pointer_cast(d_targets.data()),
      //thrust::raw_pointer_cast(d_results.data()));
  FMMTL_CUDA_CHECK;

  // Copy results back and assign
  thrust::device_ptr<result_type> d_results_ptr = thrust::device_pointer_cast(d_results);
  thrust::copy(d_results_ptr, d_results_ptr + r.size(), r.begin());

  gpu_free(d_sources);
  gpu_free(d_charges);
  gpu_free(d_targets);
  gpu_free(d_results);
}
